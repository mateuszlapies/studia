#include "hip/hip_runtime.h"
#include<stdio.h>
#include <chrono>
#include <ctime>
#define N 32
#define M 8
#define MAX 8

__global__ void f(double* x, double* out)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	double xValue = 1.0 * x[index];
	out[index] = 1.0 / (1.0 + xValue * xValue);
}

void findXs(double* tab, double wym)
{
	double max = 1.0 * MAX;
	for (int i = 0; i < wym; i++)
		tab[i] = (1.0 * i) / ((wym - 1.0) * max);
}

void zeros(double* tab, int wym) {
	for (int i = 0; i < wym; i++)
		tab[i] = 0;
}

int main(void) {
	double *x, *d_x;
	double *out, *d_out;
	int size = (N + 1) * sizeof(double);
	hipMalloc((void**)&d_x, size);
	hipMalloc((void**)&d_out, size);

	auto start = std::chrono::system_clock::now();

	x = (double*)malloc(size);
	findXs(x, (double) 1.0 * (N + 1));

	out = (double*)malloc(size);
	zeros(x, N + 1);

	double max = 1.0 * MAX;
	double n = 1.0 * N;

	double dx = max / n;

	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	f << <N / M, M >> > (d_x, d_out);
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	double sum = 0;

	for (int i = 1; i < N; i++)
	{
		sum += out[i];
	}

	sum = 4 * dx * (sum + (out[0] + out[N + 1])/2.0);
	
	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;

	printf("Out: %f | Time: %f", sum, elapsed_seconds.count());

	free(x); free(out);
	hipFree(d_x); hipFree(d_out);
	return 0;
}


