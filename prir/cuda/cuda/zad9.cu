﻿
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void action(double* x, double* out) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	double top = 1;
	for (int i = 0; i < index; i++) {
		top *= -1.0;
	}
	double bottom = 1;
	for (double s = 2 * index + 1; s > 1; s--) {
		bottom *= s;
	}

	double bottom_power = 1;
	for (int i = 0; i < 2 * index + 1; i++) {
		bottom_power *= *x;
	}

	out[index] = top / bottom *  bottom_power;
}

int main(void)
{
	double N = 3;
	double *x, *d_x;
	
	hipMalloc((void**)&d_x, sizeof(double));
	x = (double*)malloc(sizeof(double));
	*x = 0.25;

	double *out, *d_out;
	int size = N * sizeof(double);
	hipMalloc((void**)&d_out, size);
	out = (double*)malloc(size);

	hipMemcpy(d_x, x, sizeof(double), hipMemcpyHostToDevice);
	action <<<N, 8>>> (d_x, d_out);
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	double sum = 0;

	for (int i = 0; i < N; i++) {
		sum += out[i];
	}

	printf("Out: %f", sum);
	free(x); free(out);
	hipFree(d_x); hipFree(d_out);
	return 0;
}